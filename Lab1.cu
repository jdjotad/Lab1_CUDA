
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<ctype.h>
#include<math.h>
#include<time.h>
__device__ float edo_original(float t)
{
	return 9 * (powf(t, 2)) - 4 * t + 5;
}

__global__ void euler_method(float *y, float delta_t, int N)
{
	int y0 = 4;
	int tId = threadIdx.x + blockIdx.x*blockDim.x;
	float sum = 0;
	if(tId < N){
		for(int i = 0, j = 0; i <= tId ; i++, j++){ //Desde 1 / delta_t porque necesita empezar desde n=1, hasta n=10
				sum += edo_original(j*delta_t);
				y[i] = y0 + (delta_t * sum);
		}
	}
}

float edo_resuelta(float t);

int main(){
	FILE *fp;
	int i, j, N, counter = 0;
  float delta_t[6] = {powf(10, -1), powf(10, -2), powf(10, -3), powf(10, -4),
                    powf(10, -5), powf(10, -6)};
  float *y, *y_dev;
	int block_size, grid_size;
  fp = fopen("../1_b", "w");

	hipEvent_t ct1, ct2;
	float dt;
	hipEventCreate(&ct1); hipEventCreate(&ct2);

  for(j = 0 ; j < 6 ; j++)
  {
		block_size = 256; N = 10 / delta_t[j];
		grid_size = (int)ceil((float)N/ block_size);

		hipMalloc(&y_dev, sizeof(float) * N);
		y = (float*) malloc(sizeof(float) * N);

		hipEventRecord(ct1);
  	euler_method<<<grid_size,block_size>>>(y_dev, delta_t[j], N);
		hipEventRecord(ct2);
		hipMemcpy(y, y_dev, N *sizeof(float), hipMemcpyDeviceToHost);
		hipEventSynchronize(ct2);
		hipEventElapsedTime(&dt, ct1, ct2);

		fprintf(fp, "*********************************\n");
		fprintf(fp, "Con delta = %f\n", delta_t[j]);
		fprintf(fp, "*********************************\n");
  	for(i = 0 ; i < N; i++)
    {
			fprintf(fp, "t = %f\n", i+1 * delta_t[j]);
      fprintf(fp, "y[%i]=%f , %f\n", i + 1, *(y + i), edo_resuelta((i+1) * delta_t[j]));
    }
		counter++; printf("Tiempo que demora en GPU = %f [ms] para delta numero %d\n", dt, counter);
		free(y);
		hipFree(y_dev);
  }
  return 0;
}

float edo_resuelta(float t)
{
	return 3 * (powf(t, 3)) - 2 * (powf(t, 2)) + 5 * t + 4;
}
