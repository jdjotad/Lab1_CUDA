
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<ctype.h>
#include<math.h>

float* euler_method(int t0, int y0, double delta_t);


int main(){
  int t0 = 0, y0 = -1;
  double delta_t[6] = {pow(10,-1), pow(10,-2), pow(10,-3), pow(10,-4),
                    pow(10,-5), pow(10,-6)};
  float* y;
  y = euler_method(t0, y0, delta_t[0]);
  printf("%f", y[0]);
  free(y);
  return 0;
}

float* euler_method(int t0, int y0, double delta_t){
  int N = 10/delta_t;
  double sum_exp = 1;
  float* y = (float*) malloc(sizeof(float)*N); //Asignacion de memoria
  for(int i = 1; i <= N; i++){
    for(int j = 1; i<= i - 1; i++){
      sum_exp += exp(-j*delta_t);
    }
    y[i] = -1 + float(delta_t*sum_exp);
  }
  return y;
}
