
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<ctype.h>
#include<math.h>
#include<time.h>

__global__ void euler_method(float *y, float *sum, float delta_t, int N)
{
	int y0 = 4;
	int tId = threadIdx.x + blockIdx.x*blockDim.x;
	if(tId <= N){
			y[tId] = y0 + delta_t * sum[tId];
	}
}

float edo_resuelta(float t);
float edo_original(float t);
void sumatoria(float *sum, float delta_t, int N);

int main(){
	FILE *fp;
	int i, j, N, counter = 0;
  float delta_t[6] = {powf(10, -1), powf(10, -2), powf(10, -3), powf(10, -4),
                    powf(10, -5), powf(10, -6)};
  float *y, *y_dev, *sum, *sum_dev;
	int block_size, grid_size;
  fp = fopen("../1_c", "w");

	hipEvent_t ct1, ct2;
	float dt;
	hipEventCreate(&ct1); hipEventCreate(&ct2);

  for(j = 0 ; j < 6 ; j++)
  {
  	fprintf(fp, "*********************************\n");
  	fprintf(fp, "Con delta = %f\n", delta_t[j]);
  	fprintf(fp, "*********************************\n");

		block_size = 256;
		N = 10 / delta_t[j];
		grid_size = (int)ceil((float)(N +1 )/ block_size);

		hipMalloc(&y_dev, sizeof(float) * (N + 1));
		hipMalloc(&sum_dev, sizeof(float) * (N + 1));
		sum = (float*) malloc(sizeof(float) * (N + 1));
		y = (float*) malloc(sizeof(float) * (N + 1));

		sumatoria(sum, delta_t[j], N);

		hipEventRecord(ct1);
		hipMemcpy(sum_dev, sum, (N + 1)*sizeof(float), hipMemcpyHostToDevice);
  	euler_method<<<grid_size,block_size>>>(y_dev, sum_dev, delta_t[j], N);
		hipEventRecord(ct2);
		hipMemcpy(y, y_dev, (N + 1)*sizeof(float), hipMemcpyDeviceToHost);
		hipEventSynchronize(ct2);
		hipEventElapsedTime(&dt, ct1, ct2);

  	for(i = 0 ; i <= N; i++)
    {
			fprintf(fp, "%f\n", i * delta_t[j]);
      fprintf(fp, "y[%i]=%f   ,   %f\n", i, *(y + i), edo_resuelta(i * delta_t[j]));
    }
		counter++; printf("Tiempo que demora en GPU = %f [ms] para delta numero %d\n", dt, counter);
		free(y);
		free(sum);
		hipFree(y_dev);
  }
  return 0;
}

void sumatoria(float *sum, float delta_t, int N){
	sum[0] = edo_original(0);
	for(int i = 1; i <= N; i++){
		sum[i] = sum[i-1] + edo_original(i*delta_t);
	}
}

float edo_original(float t)
{
	return 9 * (powf(t, 2)) - 4 * t + 5;
}

float edo_resuelta(float t)
{
	return 3 * (powf(t, 3)) - 2 * (powf(t, 2)) + 5 * t + 4;
}
