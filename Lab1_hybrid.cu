
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<ctype.h>
#include<math.h>
#include<time.h>

__global__ void euler_method(float *y, float *sum, float delta_t, int N)
{
	int y0 = 4;
	int tId = threadIdx.x + blockIdx.x*blockDim.x;
	if(tId < N){
			y[tId] = y0 + delta_t * sum[tId];
	}
}

float edo_resuelta(float t);
float edo_original(float t);
void sumatoria(float *sum, float delta_t, int N);

int main(){
	//FILE *fp;
	//int i;
	FILE *fp_time;
	int j, N, counter = 0;
  float delta_t[6] = {powf(10, -1), powf(10, -2), powf(10, -3), powf(10, -4),
                    powf(10, -5), powf(10, -6)};
  float *y, *y_dev, *sum, *sum_dev;
	int block_size, grid_size;
  //fp = fopen("../1_c", "w");
	fp_time = fopen("1_c_time", "w");

	hipEvent_t ct1, ct2;
	float dt;
	hipEventCreate(&ct1); hipEventCreate(&ct2);

  for(j = 0 ; j < 6 ; j++)
  {
		block_size = 256;
		N = 10 / delta_t[j];
		grid_size = (int)ceil((float) N / block_size);

		hipMalloc(&y_dev, sizeof(float) * N);
		hipMalloc(&sum_dev, sizeof(float) * N);
		sum = (float*) malloc(sizeof(float) * N);
		y = (float*) malloc(sizeof(float) * N);

		sumatoria(sum, delta_t[j], N);

		hipEventRecord(ct1);
		hipMemcpy(sum_dev, sum, N *sizeof(float), hipMemcpyHostToDevice);
  	euler_method<<<grid_size,block_size>>>(y_dev, sum_dev, delta_t[j], N);
		hipEventRecord(ct2);
		hipMemcpy(y, y_dev, N *sizeof(float), hipMemcpyDeviceToHost);
		hipEventSynchronize(ct2);
		hipEventElapsedTime(&dt, ct1, ct2);
		/*
		fprintf(fp, "*********************************\n");
		fprintf(fp, "Con delta = %f\n", delta_t[j]);
		fprintf(fp, "*********************************\n");
  	for(i = 0 ; i < N; i++)
    {
			fprintf(fp, "t = %f\n", i+1 * delta_t[j]);
      fprintf(fp, "y[%i] = %f , %f\n", i + 1, *(y + i), edo_resuelta((i+1) * delta_t[j]));
    }
		*/
		counter++; printf("Tiempo que demora en HYBRID = %f [ms] para delta numero %d\n", dt, counter);
		fprintf(fp_time, "%f %f ",delta_t[j], dt);
		free(y);
		free(sum);
		hipFree(y_dev);
  }
	fclose(fp_time);
  return 0;
}

void sumatoria(float *sum, float delta_t, int N){
	sum[0] = edo_original(0);
	for(int i = 1; i < N; i++){
		sum[i] = sum[i-1] + edo_original(i*delta_t);
	}
}

float edo_original(float t)
{
	return 9 * (powf(t, 2)) - 4 * t + 5;
}

float edo_resuelta(float t)
{
	return 3 * (powf(t, 3)) - 2 * (powf(t, 2)) + 5 * t + 4;
}
