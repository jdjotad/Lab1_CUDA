
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<ctype.h>
#include<math.h>
#include<time.h>

__device__ float edo_original(float t)
{
	return 9 * (powf(t, 2)) - 4 * t + 5;
}

__global__ void euler_method_gpu(float t, float *y, float delta_t, float m)
{
	int tId = threadIdx.x + blockIdx.x*blockDim.x;
	if(tId < (int) m){
  	y[tId] = y[tId] + delta_t*(4*t - y[tId] + 3 + tId);
	}
}
void euler_method(float t, float *y, int m, float delta_t);
float edo_resuelta(float t, int j);

int main(){
  FILE *fp_time;
  float *y, *y_dev;
  clock_t start_t, end_t, total_t;
  hipEvent_t ct1, ct2;
  int j, k, counter = 0;
  int block_size, grid_size;
  float m[5] = {powf(10, 4), powf(10, 5), powf(10, 6), powf(10, 7), powf(10, 8)};
  int N = (int) powf(10,3);
  float delta_t = powf(10,-3);
  float time[15];
	float t = 0;
  float dt;

  hipEventCreate(&ct1); hipEventCreate(&ct2);
  fp_time = fopen("2_time", "w");

  for(k = 0 ; k < 5 ; k++)
  {
    // Calcular valores iniciales
    y = (float*) malloc(sizeof(float)*m[k]);
    hipMalloc(&y_dev, sizeof(float)*m[k]);
    for(j = 0; j < m[k]; j++){
      y[j] = j;
    }
    // Copiarlos a GPU
    hipMemcpy(y_dev, y, m[k]*sizeof(float), hipMemcpyHostToDevice);

    // CPU CODE
    start_t = clock();
		for(int i = 0; i < N ; i++){
			t = i*delta_t;
			euler_method(t, y, m[k], delta_t);
		}
    end_t = clock();
    total_t = end_t - start_t;
		counter++;
    // GPU CODE
    block_size = 256;
		grid_size = (int)ceil((float) m[k] / block_size);
    hipEventRecord(ct1);
		for(int i = 0; i < N ; i++){
			t = i*delta_t;
			euler_method_gpu<<<grid_size,block_size>>>(t, y_dev, delta_t, m[k]);
		}
		hipEventRecord(ct2);
		hipMemcpy(y, y_dev, m[k]*sizeof(float), hipMemcpyDeviceToHost);
		hipEventSynchronize(ct2);
		hipEventElapsedTime(&dt, ct1, ct2);
    /*
    for(j = 0; j < m[k]; j++){
      fprintf(fp, "Valor obtenido = %f , Valor real = %f j = %d\n", y[j], edo_resuelta(1,j), j+1);
    }
    */
    time[k] = (float) 1000*total_t/CLOCKS_PER_SEC;
    time[5 + k] = dt;
    //time[10 + k] = ;
    printf("Tiempo que demora en CPU = %f [ms] para m numero %d\n", ((float) 1000*total_t/CLOCKS_PER_SEC), counter);
    printf("Tiempo que demora en GPU = %f [ms] para m numero %d\n", dt, counter);
	  free(y);
    hipFree(y_dev);
  }
  for(int i = 0; i < 15; i++){
    if((i%5 == 0) && (i != 0)){
      fprintf(fp_time, "\n");
    }
    fprintf(fp_time, "%f %f ",m[i%5], time[i]);
  }
  fclose(fp_time);
  return 0;
}


void euler_method(float t, float *y, int m, float delta_t){
  for(int j = 0; j < m ; j++){
      y[j] = y[j] + delta_t*(4*t - y[j] + 3 + j);
  }
}

float edo_resuelta(float t, int j)
{
	return expf(-t) + 4*t - 1 + j;
}
